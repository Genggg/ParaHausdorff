#include "improc.h"
#include "gpukernels.h"
#include <cstdio>

using namespace cv;
using namespace std;

#define MAX_CUDA_THREADS_PER_BLOCK 32

int main(int argc, char** argv)
{
	/** Unified memory pointers */
	char* imageName = argv[1];
	Mat image;
	image = imread(imageName, 0);
	int img_rows = image.rows;
	int img_cols = image.cols;
	const int ker_rows = 5;
	const int ker_cols = 5;
	
    /** Load the image using OpenCV */
	double **src = img2Array(image);


	/** Gaussian filtering test */
	double **dst = cudaMallocManaged2D(img_rows, img_cols);
	double **gauss_kernel = getGaussianKernel(ker_rows,ker_cols,2,2);


	conv(src, img_rows, img_cols, gauss_kernel, ker_rows, ker_cols, dst);
	Mat res = array2Img(dst, img_rows, img_cols);
	imwrite( "Smoothed_Image.jpg", res);

    /** Distance transform test */
	double **dst1 = cudaMallocManaged2D(img_rows, img_cols);
    distTrans(src, img_rows, img_cols ,dst1);
    Mat res1 = array2Img(dst1, img_rows, img_cols);
	imwrite( "cell_distance.jpg", res1);

	/** Image dilation test */
	double **dst2 = cudaMallocManaged2D(img_rows, img_cols);
	dilate(dst1, img_rows, img_cols, 2, dst2);
	Mat res2 = array2Img(dst2, img_rows, img_cols);
	imwrite( "dilated_cell.jpg", res2);
	
	/** Test on GPU Global Gaussian Filtering Kernel */ 

	// Show some related infomation regarding the GPU

		int deviceCount;
		hipGetDeviceCount(&deviceCount);
		if (deviceCount == 0) {
			fprintf(stderr, "error: no devices supporting CUDA.\n");
			exit(EXIT_FAILURE);
		}
		int dev = 0;
		hipSetDevice(dev);
	
		hipDeviceProp_t devProps;
		if (hipGetDeviceProperties(&devProps, dev) == 0)
		{
			printf("Using device %d:\n", dev);
			printf("%s; Global Memory: %fGB; Shared Memory/block: %lu KB; Compute v%d.%d; Clock: %f GHz\n",
				devProps.name, (float)devProps.totalGlobalMem / (1024*1024*1024), 
				devProps.sharedMemPerBlock/(1024), (int)devProps.major, (int)devProps.minor, 
				(float)devProps.clockRate/(1000*1000));
		}

	
	double **dstg = cudaMallocManaged2D(img_rows, img_cols);

	// Assign the number of blocks
	const unsigned num_threads_row = MAX_CUDA_THREADS_PER_BLOCK;
	const unsigned num_threads_col = MAX_CUDA_THREADS_PER_BLOCK;
	// Block dim: 32 x 32
	const unsigned num_blocks_row = (img_rows + num_threads_row) / num_threads_row;
	const unsigned num_blocks_col = (img_cols + num_threads_col) / num_threads_col;

	const dim3 num_blocks (num_blocks_col, num_blocks_row);
	const dim3 num_threads (num_threads_col, num_threads_row);

	hipEvent_t start, stop;
    hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipEventRecord(start, 0);
	// No shared memory
	convGPUGlobal<<< num_blocks, num_threads >>>
	 (src, img_rows, img_cols, gauss_kernel, ker_rows, ker_cols, dstg);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	fprintf(stdout, "Done Gaussian-Global on GPU.\n");
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop); 
	fprintf(stdout, "Time elapsed: %f ms\n", elapsedTime);

	// Transfer the output to the CPU
	fprintf(stdout, "Memory copy done.\n");
	cv::Mat resg = array2Img(dstg, img_rows, img_cols);
	imwrite( "Smoothed_Image_GPU.jpg", resg);


	return 0;											
}
